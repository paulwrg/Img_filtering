#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <unistd.h>
#include <limits.h>
#include "hip/hip_runtime.h"
#include "filter_cuda.h"

#define debug 0

int print_time = 0;
int blur_size = 5; 
int threshold = 20;

__global__ void gray(pixel *im, int height, int width)
{
  int moy, pos;
  pos = threadIdx.x + blockIdx.x * blockDim.x;

  if (pos < width * height)
  {
    moy = (im[pos].r + im[pos].g + im[pos].b) / 3;
    if (moy < 0)
      moy = 0;
    if (moy > 255)
      moy = 255;

    im[pos].r = moy;
    im[pos].g = moy;
    im[pos].b = moy;
  }
}

__global__ void blur(pixel *im, pixel *im_new, int *end,
                     int height, int width, int size, int threshold)
{
  /* Perform one blur iteration and store in end if we need more */

  int j, k, pos;

  pos = threadIdx.x + blockIdx.x * blockDim.x;
  j = pos / width;
  k = pos % width;

  if (k == 0 && j == 0)
  {
    /* One process only in charge of updating end */
    *end = 1;
  }

  if (k >= size && k < width - size)
  {

    if (j >= size && j < height / 10 - size || j >= height * 0.9 + size && j < height - size)
    {
      /* If in the top or bottom 10% :
         Apply blur on top or bottom part of image (10%) */
      int stencil_j, stencil_k;
      int t_r = 0;
      int t_g = 0;
      int t_b = 0;

      for (stencil_j = -size; stencil_j <= size; stencil_j++)
      {
        for (stencil_k = -size; stencil_k <= size; stencil_k++)
        {
          t_r += im[CONV(j + stencil_j, k + stencil_k, width)].r;
          t_g += im[CONV(j + stencil_j, k + stencil_k, width)].g;
          t_b += im[CONV(j + stencil_j, k + stencil_k, width)].b;
        }
      }

      im_new[CONV(j, k, width)].r = t_r / ((2 * size + 1) * (2 * size + 1));
      im_new[CONV(j, k, width)].g = t_g / ((2 * size + 1) * (2 * size + 1));
      im_new[CONV(j, k, width)].b = t_b / ((2 * size + 1) * (2 * size + 1));
    }

    if (j >= height / 10 - size && j < height * 0.9 + size)
    {
      /* Just copy the middle part of the image */
      im_new[CONV(j, k, width)].r = im[CONV(j, k, width)].r;
      im_new[CONV(j, k, width)].g = im[CONV(j, k, width)].g;
      im_new[CONV(j, k, width)].b = im[CONV(j, k, width)].b;
    }
  }

  // Wait until all threads have written in the memory
  __threadfence();

  // Test the end condition
  if (j >= 1 && j < height - 1 && k >= 1 && k < width - 1)
  {
    float diff_r;
    float diff_g;
    float diff_b;

    diff_r = (im_new[CONV(j, k, width)].r - im[CONV(j, k, width)].r);
    diff_g = (im_new[CONV(j, k, width)].g - im[CONV(j, k, width)].g);
    diff_b = (im_new[CONV(j, k, width)].b - im[CONV(j, k, width)].b);

    if (diff_r > threshold || -diff_r > threshold ||
        diff_g > threshold || -diff_g > threshold ||
        diff_b > threshold || -diff_b > threshold)
    {
      *end = 0;
    }
  }

  // Wait for all the threads to have tested the end condition
  __threadfence();

  if (j >= 1 && j < height - 1 && k >= 1 && k < width - 1)
  {
    // Erase and copy for new iteration
    im[CONV(j, k, width)].r = im_new[CONV(j, k, width)].r;
    im[CONV(j, k, width)].g = im_new[CONV(j, k, width)].g;
    im[CONV(j, k, width)].b = im_new[CONV(j, k, width)].b;
  }
}

__global__ void sobel(pixel *im, pixel *im_new, int height, int width)
{
  int i, j, pos;

  pos = threadIdx.x + blockIdx.x * blockDim.x;
  i = pos / width;
  j = pos % width;

  int pixel_blue_no, pixel_blue_n, pixel_blue_ne;
  int pixel_blue_so, pixel_blue_s, pixel_blue_se;
  int pixel_blue_o, pixel_blue_e;

  float deltaX_blue;
  float deltaY_blue;
  float val_blue;

  if (i >= 1 && i < height - 1 && j >= 1 && j < width - 1)
  {
    pixel_blue_no = im[CONV(i - 1, j - 1, width)].b;
    pixel_blue_n = im[CONV(i - 1, j, width)].b;
    pixel_blue_ne = im[CONV(i - 1, j + 1, width)].b;
    pixel_blue_so = im[CONV(i + 1, j - 1, width)].b;
    pixel_blue_s = im[CONV(i + 1, j, width)].b;
    pixel_blue_se = im[CONV(i + 1, j + 1, width)].b;
    pixel_blue_o = im[CONV(i, j - 1, width)].b;
    pixel_blue_e = im[CONV(i, j + 1, width)].b;

    deltaX_blue = -pixel_blue_no + pixel_blue_ne - 2 * pixel_blue_o + 2 * pixel_blue_e - pixel_blue_so + pixel_blue_se;
    deltaY_blue = pixel_blue_se + 2 * pixel_blue_s + pixel_blue_so - pixel_blue_ne - 2 * pixel_blue_n - pixel_blue_no;
    val_blue = sqrt(deltaX_blue * deltaX_blue + deltaY_blue * deltaY_blue) / 4;

    if (val_blue > 50)
    {
      im_new[CONV(i, j, width)].r = 255;
      im_new[CONV(i, j, width)].g = 255;
      im_new[CONV(i, j, width)].b = 255;
    }
    else
    {
      im_new[CONV(i, j, width)].r = 0;
      im_new[CONV(i, j, width)].g = 0;
      im_new[CONV(i, j, width)].b = 0;
    }
  }

  else
  {
    if (i < height && j < width)
    {
      im_new[CONV(i, j, width)] = im[CONV(i, j, width)];
    }
  }
}

void apply_all_filters_gpu(animated_gif *image)
{
  /** 
    * Apply the three last filters with the help of the GPU.
    * To avoid memcopying 3 times, we merged apply_gray, apply_blur and apply_sobel
    **/

  int im_num;
  int width = image->width[0];
  int height = image->height[0];
  int size = width * height;

  int *end_dev, end_host; // to know when blur has finished

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);

  /**
     * Allocation on the device once for all the images (if multiple)
     * Memory allocation + dimension of grid
     **/
  dim3 dimGrid(10*size / deviceProp.maxThreadsPerBlock + 1);
  dim3 dimBlock(deviceProp.maxThreadsPerBlock/10);

  pixel *device_image, *device_new;
  hipMalloc(&device_image, size * sizeof(pixel));
  hipMalloc(&device_new, size * sizeof(pixel));
  hipMalloc(&end_dev, sizeof(int));
#if debug
  printf("420\n");
#endif  
  /* For all images, blur than sobel */
  for (im_num = 0; im_num < image->n_images; im_num++)
  {
    hipMemcpy(device_image, image->p[im_num], size * sizeof(pixel), hipMemcpyHostToDevice);

    gray<<<dimGrid, dimBlock>>>(device_image, height, width);
    /* Bluring while it isn't finished */
    int num_iter = 0;
    end_host = 1;
    do
    {
      num_iter++;
      blur<<<dimGrid, dimBlock>>>(device_image, device_new, end_dev, height, width, blur_size, threshold);
      hipMemcpy(&end_host, end_dev, sizeof(int), hipMemcpyDeviceToHost);
    } while (threshold > 0 && !end_host);

    /* Applying sobel */
    sobel<<<dimGrid, dimBlock>>>(device_image, device_new, height, width);

    hipMemcpy(image->p[im_num], device_new, size * sizeof(pixel), hipMemcpyDeviceToHost);
#if debug
    printf("445\n");
#endif 
  }

  hipFree(device_image);
  hipFree(device_new);
  hipFree(end_dev);
}